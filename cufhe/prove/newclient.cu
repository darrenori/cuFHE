// Client side C/C++ program to demonstrate Socket
// programming
#include <arpa/inet.h>
<<<<<<< HEAD
#include <stdlib.h>
#include <arpa/inet.h>
=======
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
>>>>>>> bc36fd060f54999ccb8fe41a0bed7a8b29624f46
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
<<<<<<< HEAD
#define SIZE 1024
=======
#define SIZE 500000
>>>>>>> bc36fd060f54999ccb8fe41a0bed7a8b29624f46

  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;
void send_file(FILE *fp, int sockfd){
  int n;
  char data[SIZE] = {0};

  while(fgets(data, SIZE, fp) != NULL) {
    if (send(sockfd, data, sizeof(data), 0) == -1) {
      perror("[-]Error in sending file.");
      exit(1);
    }
    bzero(data, SIZE);
  }
}



void send_file(FILE *fp, int sockfd){
  int n;
  char data[SIZE] = {0};

  while(fgets(data, SIZE, fp) != NULL) {
    if (send(sockfd, data, sizeof(data), 0) == -1) {
      perror("[-]Error in sending file.");
      exit(1);
    }
    bzero(data, SIZE);
  }
}


void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

int main(int argc, char const* argv[])
{
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    //SetSeed();


    PriKey pri_key; // private key
    bool correct = true;

    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();


    for (int i = 0; i < numBits; i ++) {
 	pt[i] = 0;
      	pt1[i] = 0;
   	Encrypt(ct[i], pt[i], pri_key);
      	Encrypt(ct1[i], pt1[i], pri_key);
    }


    Synchronize();





/*
    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
    }

    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher1/ct" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }


<<<<<<< HEAD

    // Change this IP
  char *ip = "69.69.69.1";
  // Change this host port
  int port = 4380;
  int e;

  int sockfd;
  struct sockaddr_in server_addr;
  FILE *fp;
  // Change this file name and file path if you need
  char *filename = "send.txt";

  sockfd = socket(AF_INET, SOCK_STREAM, 0);
  if(sockfd < 0) {
    perror("[-]Error in socket");
    exit(1);
  }
  printf("[+]Server socket created successfully.\n");
 
  server_addr.sin_family = AF_INET;
  server_addr.sin_port = port;
  server_addr.sin_addr.s_addr = inet_addr(ip);

  e = connect(sockfd, (struct sockaddr*)&server_addr, sizeof(server_addr));
  if(e == -1) {
    perror("[-]Error in socket");
    exit(1);
  }
        printf("[+]Connected to Server.\n");

  fp = fopen(filename, "r");
  if (fp == NULL) {
    perror("[-]Error in reading file.");
    exit(1);
  }

  send_file(fp, sockfd);
  printf("[+]File data sent successfully.\n");

=======
    remove("cipher/overall");
    for (int i=0; i< numBits; i++) {
	    std::ifstream if_a("cipher/ct"+std::to_string(i), std::ios_base::app);
	    std::ofstream of_a("cipher/overall", std::ios_base::app);
	    of_a << if_a.rdbuf();
    }
    for (int i=0; i< numBits; i++) {
	    std::ifstream if_a("cipher1/ct"+std::to_string(i), std::ios_base::app);
	    std::ofstream of_a("cipher/overall", std::ios_base::app);
	    of_a << if_a.rdbuf();
    }
*/

/*

    // Change this IP
  char *ip = "69.69.69.1";
  // Change this host port
  int port = 4380;
  int e;

  int sockfd;
  struct sockaddr_in server_addr;
  FILE *fp;
  // Change this file name and file path if you need
  char *filename = "cipher/overall";

  sockfd = socket(AF_INET, SOCK_STREAM, 0);
  if(sockfd < 0) {
    perror("[-]Error in socket");
    exit(1);
  }
  printf("[+]Server socket created successfully.\n");

  server_addr.sin_family = AF_INET;
  server_addr.sin_port = port;
  server_addr.sin_addr.s_addr = inet_addr(ip);

  e = connect(sockfd, (struct sockaddr*)&server_addr, sizeof(server_addr));
  if(e == -1) {
    perror("[-]Error in socket");
    exit(1);
  }
        printf("[+]Connected to Server.\n");

  fp = fopen(filename, "r");
  if (fp == NULL) {
    perror("[-]Error in reading file.");
    exit(1);
  }

  send_file(fp, sockfd);
  printf("[+]File data sent successfully.\n");

>>>>>>> bc36fd060f54999ccb8fe41a0bed7a8b29624f46
        printf("[+]Closing the connection.\n");
  close(sockfd);




  */


    //-------------------READING BACK DATA FROM SERVER----------------------//
    for (int i = 0; i < numBits; i ++) {
            string filename = "cipherRes/ct" + std::to_string(i);
            ReadCtxtFromFile(ctRes[i],filename);
    }


    //READ COMPUTED DATA FROM SERVER HERE!
    int cnt_failures = 0;
    for (int i = 0; i < numBits; i ++) {
      NandCheck(ptRes[i], pt[i], pt1[i]);
      Decrypt(pt1[i], ctRes[i], pri_key);
      if (pt1[i].message_ != ptRes[i].message_) {
        std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
        correct = false;
        cnt_failures += 1;
        //std::cout<< "Fail at iteration: " << i <<std::endl;
      }
    }


    if (correct)
      cout<< "PASS" <<endl;
    else
      cout<< "FAIL:\t" << cnt_failures << "/" << numBits <<endl;
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;


}
