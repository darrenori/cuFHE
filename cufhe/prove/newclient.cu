// Client side C/C++ program to demonstrate Socket
// programming
#include <arpa/inet.h>
#include <stdlib.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>

  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;





void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

int main(int argc, char const* argv[])
{
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    //SetSeed();


    PriKey pri_key; // private key
    bool correct = true;

    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();


    for (int i = 0; i < numBits; i ++) {
 	pt[i] = 0;
      	pt1[i] = 0;
   	Encrypt(ct[i], pt[i], pri_key);
      	Encrypt(ct1[i], pt1[i], pri_key);
    }


    Synchronize();






    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
    }

    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher1/ct" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }


    remove("cipher/overall");
    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }







 

    return 0;

    //-------------------READING BACK DATA FROM SERVER----------------------//
    for (int i = 0; i < numBits; i ++) {
            string filename = "cipherRes/ct" + std::to_string(i);
            ReadCtxtFromFile(ctRes[i],filename);
    }


    //READ COMPUTED DATA FROM SERVER HERE!
    int cnt_failures = 0;
    for (int i = 0; i < numBits; i ++) {
      NandCheck(ptRes[i], pt[i], pt1[i]);
      Decrypt(pt1[i], ctRes[i], pri_key);
      if (pt1[i].message_ != ptRes[i].message_) {
        std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
        correct = false;
        cnt_failures += 1;
        //std::cout<< "Fail at iteration: " << i <<std::endl;
      }
    }


    if (correct)
      cout<< "PASS" <<endl;
    else
      cout<< "FAIL:\t" << cnt_failures << "/" << numBits <<endl;
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;


}
