#include "hip/hip_runtime.h"
// Client side C/C++ program to demonstrate Socket
// programming
#include <stdlib.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
#include <netinet/in.h>

#include <ios>

  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;



class Client_socket{
    fstream file;

    int PORT;
    
    int general_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:
        Client_socket(){
	
	};

	void start_everything(int number){
            create_socket();
            PORT = number;

	    cout << "The port is " << PORT << "\n";

            address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);
            if(inet_pton(AF_INET, "69.69.69.1", &address.sin_addr)<=0) { 
                cout<<"[ERROR] : Invalid address\n";
            }

            create_connection();
            
            file.open("cipher/overall", ios::out | ios::trunc | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }
        };

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) < 0) { 
                perror("[ERROR] : Socket failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
        }

        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) { 
                perror("[ERROR] : connection attempt failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connection Successfull.\n";
        }

        void close_socket(){
	    close(general_socket_descriptor);
	};

        void receive_file(){
            char buffer[2200024] = {};
	    bzero(buffer, sizeof(buffer));
	    int count = 0;
	    while(1){

	            int valread = read(general_socket_descriptor , buffer, 2200024);
		    if(valread == 0)
			    break;
		    file<<buffer;
		    bzero(buffer, sizeof(buffer));
	    };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
	    file.close();
	    close(general_socket_descriptor);
        }

	void split_file(){
            int count = 0;
	    std::ifstream file("rec.txt");
	    
	    std::string filenames[64];
            for (int i = 0; i < 64; i ++){
                string filename = "Ctxt" + std::to_string(i);
		filenames[i] = filename;
	    };

	    if (file.is_open()) {
    	 	std::string line;
    		while (std::getline(file, line)) {
		      int fileChoice = floor(count/501);
	              ofstream Myfile;
		      Myfile.open(filenames[fileChoice], fstream::app);
		      Myfile << line.c_str() << endl;
		      count += 1; 
	        };
	     };
        };
 
        void transmit_file(){

	    fstream file2;
            
            printf("============================\n");	    
	    file2.open("cipher/overall", ios::in | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Send File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            	    
            std::string contents((std::istreambuf_iterator<char>(file2)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";	
	}

};


void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

int main(int argc, char const* argv[])
{
    int port1 = 4380;
    int port2 = 4381;
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    //SetSeed();


    PriKey pri_key; // private key
    bool correct = true;

    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();


    for (int i = 0; i < numBits; i ++) {
 	pt[i] = 0;
      	pt1[i] = 0;
   	Encrypt(ct[i], pt[i], pri_key);
      	Encrypt(ct1[i], pt1[i], pri_key);
    }


    Synchronize();





    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
	    filename = "cipher1/ct" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }

    remove("cipher/overall");
    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }

    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher1/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }

    Client_socket C;
    C.start_everything(port1);
    C.transmit_file();
    C.close_socket();


    return 0;


    //-------------------READING BACK DATA FROM SERVER----------------------//
    //@RUSSEL HERE
    for (int i = 0; i < numBits; i ++) {
            string filename = "cipherRes/ct" + std::to_string(i);
            ReadCtxtFromFile(ctRes[i],filename);
    }


    //READ COMPUTED DATA FROM SERVER HERE!
    int cnt_failures = 0;
    for (int i = 0; i < numBits; i ++) {
      NandCheck(ptRes[i], pt[i], pt1[i]);
      Decrypt(pt1[i], ctRes[i], pri_key);
      if (pt1[i].message_ != ptRes[i].message_) {
        std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
        correct = false;
        cnt_failures += 1;
        //std::cout<< "Fail at iteration: " << i <<std::endl;
      }
    }


    if (correct)
      cout<< "PASS" <<endl;
    else
      cout<< "FAIL:\t" << cnt_failures << "/" << numBits <<endl;
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;


}
