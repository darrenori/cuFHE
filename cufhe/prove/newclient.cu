// Client side C/C++ program to demonstrate Socket
// programming
#include <arpa/inet.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>

#define PORT 4380
  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

int main(int argc, char const* argv[])
{
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    SetSeed();


    PriKey pri_key; // private key
    bool correct = true;

    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


    Ptxt* pt = new Ptxt[numBits * 2];
    Ptxt* pt1 = new Ptxt[numBits * 2];
    Ptxt* ptRes = new Ptxt[numBits * 2];
    Ctxt* ct = new Ctxt[numBits * 2];
    Ctxt* ct1 = new Ctxt[numBits * 2];
    Ctxt* ctRes = new Ctxt[numBits * 2];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();


    for (int i = 0; i < numBits; i ++) {
      //pt[i] = rand() % Ptxt::kPtxtSpace;
      pt[i] = 0;
      Encrypt(ct[i], pt[i], pri_key);
    }

    for (int i = 0; i < numBits; i ++) {
      //pt1[i] = rand() % Ptxt::kPtxtSpace;
      pt1[i] = 0;
      Encrypt(ct1[i], pt1[i], pri_key);
    }

    Synchronize();















    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
    }

    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher1/ct" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }


    // End of Generation of CT
    char buffer[1024] = { 0 };
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0) {
        printf("\n Socket creation error \n");
        return -1;
    }
  
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_port = htons(PORT);
  
    // Convert IPv4 and IPv6 addresses from text to binary
    // form
    if (inet_pton(AF_INET, "69.69.69.1", &serv_addr.sin_addr)
        <= 0) {
        printf(
            "\nInvalid address/ Address not supported \n");
        return -1;
    }
  
    if ((client_fd
         = connect(sock, (struct sockaddr*)&serv_addr,
                   sizeof(serv_addr)))
        < 0) {
        printf("\nConnection Failed \n");
        return -1;
    }


    std::string s = std::to_string(23);
    char const *pchar = s.c_str(); 

    send(sock, pchar, strlen(pchar), 0);
    printf("Sending to server...\n");
    valread = read(sock, buffer, 1024);
    printf("%s\n", buffer);
  
    // closing the connected socket
    close(client_fd);
    



















    //-------------------READING BACK DATA FROM SERVER----------------------//
    for (int i = 0; i < numBits; i ++) {
            string filename = "cipherresult/ct" + std::to_string(i);
            ReadCtxtFromFile(ctRes[i],filename);
    }


    //READ COMPUTED DATA FROM SERVER HERE!

    int cnt_failures = 0;
    for (int i = 0; i < numBits; i ++) {
      NandCheck(ptRes[i], pt[i], pt1[i]);
      Decrypt(pt1[i], ctRes[i], pri_key);
      if (pt1[i].message_ != ptRes[i].message_) {
        std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
        correct = false;
        cnt_failures += 1;
        //std::cout<< "Fail at iteration: " << i <<std::endl;
      }
    }


    if (correct)
      cout<< "PASS" <<endl;
    else
      cout<< "FAIL:\t" << cnt_failures << "/" << numBits <<endl;
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;


}
