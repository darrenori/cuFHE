// Client side C/C++ program to demonstrate Socket
// programming
#include <arpa/inet.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>

#define PORT 4380
  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;


int main(int argc, char const* argv[])
{
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
 //   uint32_t kNumTests = kNumSMs * 32;// * 8;
    uint32_t kNumLevels = 4;
    int numBits = 32;

    SetSeed();


    PriKey pri_key; // private key
    PubKey pub_key; // public key
    bool correct = true;

    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");
    ReadPubKeyFromFile(pub_key,"finalkeys/publickey1.txt");


    Ptxt* pt = new Ptxt[numBits * 2];
    Ptxt* pt1 = new Ptxt[numBits * 2];
    Ptxt* ptRes = new Ptxt[numBits * 2];
    Ctxt* ct = new Ctxt[numBits * 2];
    Ctxt* ct1 = new Ctxt[numBits * 2];
    Ctxt* ctRes = new Ctxt[numBits * 2];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();


    for (int i = 0; i < numBits; i ++) {
      //pt[i] = rand() % Ptxt::kPtxtSpace;
      pt[i] = 0;
      pt[2]=1;
      Encrypt(ct[i], pt[i], pri_key);
    }

    for (int i = 0; i < numBits; i ++) {
      //pt1[i] = rand() % Ptxt::kPtxtSpace;
      pt1[i] = 0;
      pt1[2]=1;
      Encrypt(ct1[i], pt1[i], pri_key);
    }

    Synchronize();

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
    }

    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher1/ct1" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }



    // End of Generation of CT
    char buffer[1024] = { 0 };
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0) {
        printf("\n Socket creation error \n");
        return -1;
    }
  
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_port = htons(PORT);
  
    // Convert IPv4 and IPv6 addresses from text to binary
    // form
    if (inet_pton(AF_INET, "69.69.69.1", &serv_addr.sin_addr)
        <= 0) {
        printf(
            "\nInvalid address/ Address not supported \n");
        return -1;
    }
  
    if ((client_fd
         = connect(sock, (struct sockaddr*)&serv_addr,
                   sizeof(serv_addr)))
        < 0) {
        printf("\nConnection Failed \n");
        return -1;
    }


    std::string s = std::to_string(23);
    char const *pchar = s.c_str(); 

    send(sock, pchar, strlen(pchar), 0);
    printf("Sending to server...\n");
    valread = read(sock, buffer, 1024);
    printf("%s\n", buffer);
  
    // closing the connected socket
    close(client_fd);
    return 0;
    
}
