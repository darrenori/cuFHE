// Client side C/C++ program to demonstrate Socket
// programming
#include <stdlib.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
#include <netinet/in.h>

#include <ios>

  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;




class Server_socket{
    fstream file;

    int PORT;

    int general_socket_descriptor;
    int new_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:
        Server_socket(){
            create_socket();
            PORT = 8050;

            address.sin_family = AF_INET;
            address.sin_addr.s_addr = INADDR_ANY;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

            bind_socket();
            set_listen_set();
            accept_connection();

            file.open("cipher/overall", ios::in | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }
        }


	void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) == 0) {
                perror("[ERROR] : Socket failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
        }

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))<0) {
                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void accept_connection(){
            if ((new_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void transmit_file(){
            std::string contents((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(new_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";
        }
};


void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

int main(int argc, char const* argv[])
{
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    //SetSeed();


    PriKey pri_key; // private key
    bool correct = true;

    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();


    for (int i = 0; i < numBits; i ++) {
 	pt[i] = 0;
      	pt1[i] = 0;
   	Encrypt(ct[i], pt[i], pri_key);
      	Encrypt(ct1[i], pt1[i], pri_key);
    }


    Synchronize();





    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
	    filename = "cipher1/ct" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }

    remove("cipher/overall");
    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }

    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher1/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }


    Server_socket S;
    S.transmit_file();

    return 0;


    //-------------------READING BACK DATA FROM SERVER----------------------//
    //@RUSSEL HERE
    for (int i = 0; i < numBits; i ++) {
            string filename = "cipherRes/ct" + std::to_string(i);
            ReadCtxtFromFile(ctRes[i],filename);
    }


    //READ COMPUTED DATA FROM SERVER HERE!
    int cnt_failures = 0;
    for (int i = 0; i < numBits; i ++) {
      NandCheck(ptRes[i], pt[i], pt1[i]);
      Decrypt(pt1[i], ctRes[i], pri_key);
      if (pt1[i].message_ != ptRes[i].message_) {
        std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
        correct = false;
        cnt_failures += 1;
        //std::cout<< "Fail at iteration: " << i <<std::endl;
      }
    }


    if (correct)
      cout<< "PASS" <<endl;
    else
      cout<< "FAIL:\t" << cnt_failures << "/" << numBits <<endl;
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;


}
