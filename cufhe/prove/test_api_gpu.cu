/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

void NotCheck(Ptxt& out, const Ptxt& in) {
	out.message_ = (~in.message_) & 0x1;
}

void CopyCheck(Ptxt& out, const Ptxt& in){
	out.message_ = in.message_;
}

void addBits(Ctxt *r, Ctxt &a, Ctxt &b, Ctxt *carry) {
	Ctxt *t1 = new Ctxt[1];
    Ctxt *t2 = new Ctxt[1];
	Xor(t1[0], a, carry[0]);
    Xor(t2[0], b, carry[0]);
	Synchronize();
	Xor(r[0], a, t2[0]);
	And(t1[0], t1[0], t2[0]);
	Synchronize();
	Xor(r[1], carry[0], t1[0]);
	Synchronize();
	delete [] t1;
	delete [] t2;
}

void addNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
	
	Ctxt *carry = new Ctxt[1];
    Ctxt *bitResult = new Ctxt[2];

	Xor(ctRes[0], ctA[0], ctB[0]);
	And(carry[0], ctA[0], ctB[0]);
	Synchronize();
	for(int i = 1; i < nBits; i++) {
		addBits(bitResult, ctA[i], ctB[i], carry);
		Copy(ctRes[i], bitResult[0]);
		Copy(carry[0], bitResult[1]);
		Synchronize();
	}
	delete [] carry;
	delete [] bitResult;
}


int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  uint32_t kNumTests = kNumSMs * 32;// * 8;
  uint32_t kNumLevels = 4;
  uint32_t val1 = 1;
  uint32_t val2 = 2;
  int numBits = 32;

  SetSeed(); // set random seed

  PriKey pri_key; // private key
  PubKey pub_key; // public key

  ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");
  ReadPubKeyFromFile(pub_key,"finalkeys/publickey1.txt");

  Ptxt* pt = new Ptxt[numBits * 2];
  Ptxt* pt1 = new Ptxt[numBits * 2];
  Ptxt* ptRes = new Ptxt[numBits * 2];
  Ctxt* ct = new Ctxt[numBits * 2];
  Ctxt* ct1 = new Ctxt[numBits * 2];
  Ctxt* ctRes = new Ctxt[numBits * 2];
  Synchronize();
  bool correct;

  cout<< "------ Key Generation ------" <<endl;
  KeyGen(pub_key, pri_key);

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing


  cout<< "Number of tests:\t" << numBits <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();


  correct = true;
  for (int i = 0; i < numBits; i ++) {
    //pt[i] = rand() % Ptxt::kPtxtSpace;
    pt[i] = 0;
    pt[2]=1;
    Encrypt(ct[i], pt[i], pri_key);
  }

  for (int i = 0; i < numBits; i ++) {
    //pt1[i] = rand() % Ptxt::kPtxtSpace;
    pt1[i] = 0;
    pt1[2]=1;
    Encrypt(ct1[i], pt1[i], pri_key);
  }

 // std::string abc="abc.txt";
 // WriteCtxtToFile(ct[0],abc);

  Synchronize();

  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  // Here, pass streams to gates for parallel gates.
  cout<< "------ Test NAND Gate ------" <<endl;
  for (int i = 0; i < numBits; i ++) {
    Nand(ctRes[i], ct[i], ct1[i], st[i % kNumSMs]);
  }



  Synchronize();
  

//  addNumbers(ctRes, ct, ct1, numBits);


  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&et, start, stop);
  cout<< et / numBits / kNumLevels << " ms / gate" <<endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);
 /* for (int i =0; i < numBits; i ++){
	Decrypt(pt1[i], ctRes[i], pri_key);
	cout << pt1[i].message_;
  }*/

  int cnt_failures = 0;
  for (int i = 0; i < numBits; i ++) {
    NandCheck(ptRes[i], pt[i], pt1[i]);
    Decrypt(pt1[i], ctRes[i], pri_key);
    if (pt1[i].message_ != ptRes[i].message_) {
      std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
      correct = false;
      cnt_failures += 1;
      //std::cout<< "Fail at iteration: " << i <<std::endl;
    }
  }


 
  if (correct)
    cout<< "PASS" <<endl;
  else
    cout<< "FAIL:\t" << cnt_failures << "/" << numBits <<endl;
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  
  delete [] st;

  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  return 0;
}
