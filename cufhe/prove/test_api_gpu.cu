#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include<iostream>
#include<fstream>
#include<stdio.h>
#include <unistd.h>
#include <sys/socket.h>
#include <stdlib.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <strings.h>
#include <string>
#include <cmath>
#include <cstring>
#include <include/cufhe_gpu.cuh>
#include<bits/stdc++.h>

using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

void NotCheck(Ptxt& out, const Ptxt& in) {
	out.message_ = (~in.message_) & 0x1;
}

void CopyCheck(Ptxt& out, const Ptxt& in){
	out.message_ = in.message_;
}


void addBits(Ctxt *r, Ctxt &a, Ctxt &b, Ctxt *carry) {
	Ctxt *t1 = new Ctxt[1];
    Ctxt *t2 = new Ctxt[1];
	Xor(t1[0], a, carry[0]);
    Xor(t2[0], b, carry[0]);
	Synchronize();
	Xor(r[0], a, t2[0]);
	And(t1[0], t1[0], t2[0]);
	Synchronize();
	Xor(r[1], carry[0], t1[0]);
	Synchronize();
	delete [] t1;
	delete [] t2;
}

void addNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
  	
	Ctxt *carry = new Ctxt[1];
        Ctxt *bitResult = new Ctxt[2];

	Xor(ctRes[nBits-1], ctA[nBits-1], ctB[nBits-1]);
	And(carry[0], ctA[nBits-1], ctB[nBits-1]);
	Synchronize();
	for(int i = nBits-2; i > 0; i--) {
		addBits(bitResult, ctA[i], ctB[i], carry);
		Copy(ctRes[i], bitResult[0]);
		Copy(carry[0], bitResult[1]);
		Synchronize();
	}
	Copy(ctRes[0], carry[0]);
	Synchronize();
	delete [] carry;
	delete [] bitResult;
}



void twoComplements(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, Ctxt *minusEnd, int nBits){
             
	    Ctxt *twoRes = new Ctxt[nBits];
	
	    // Inverse B
            for(int i = 0; i < nBits; i++){
                Not(ctB[i], ctB[i]);
            }
 
            Synchronize();
 
            // Add One to B
            addNumbers(twoRes, minusEnd, ctB, nBits);
 
            // Add result to A
            addNumbers(ctRes, ctA, twoRes, nBits);

       	    Not(ctRes[0], ctRes[0]);

	    delete [] twoRes;

};

void subNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
	Ctxt *minusEnd = new Ctxt[nBits];
	
	for(int i = 0; i < nBits; i ++){
	    Copy(minusEnd[i], ctA[0]);
	};

	Not(minusEnd[nBits-1], minusEnd[nBits-1]);

        twoComplements(ctRes, ctA, ctB, minusEnd, nBits);

	delete [] minusEnd;
};

void mulNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int iBits, int oBits){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	uint32_t kNumSMs = prop.multiProcessorCount;
	Stream* st = new Stream[kNumSMs];
	for (int i = 0; i < kNumSMs; i ++) {
		st[i].Create();
	}

	Ctxt* tempSum = new Ctxt[oBits];
	Ctxt* tempSum2 = new Ctxt[oBits];
	Ctxt* andRes = new Ctxt[iBits];
	Ctxt* empty = new Ctxt[oBits];

	//MAKE IT ZERO
	Ctxt* zero = new Ctxt[1];
	And(zero[0],ctA[0],ctB[0],st[0 % kNumSMs]);
	Xor(zero[0],zero[0],zero[0],st[0 % kNumSMs]);
	Synchronize();

	for(int i=0; i<oBits; i++){
		Copy(tempSum[i],zero[0]);
		Copy(tempSum2[i],zero[0]);
	};

	int co=0;
	int counter=0;
	Synchronize();

	for(int i = iBits-1; i > -1; i--) {
		co=0;
		co=counter;

		Ctxt* andResLeft = new Ctxt[oBits];
		//initalize nresleft to be 'nothing'
		for(int i=0; i<oBits; i++){
			Copy(andResLeft[i],zero[0]);
		}

		for(int j = 0; j < iBits; j++) {
			And(andRes[j], ctA[oBits-1-j], ctB[oBits-1-counter], st[j % kNumSMs]);
		}
		Synchronize();

		for(int j = 0; j < iBits; j++) {
			//cout << oBits-1-co;
			Copy(andResLeft[oBits-1-co], andRes[j]);
			co++;
		}

		Synchronize();

                if(counter==0) {
			addNumbers(tempSum, andResLeft, tempSum2, oBits);
			Synchronize();
		} else {
			addNumbers(tempSum, andResLeft, tempSum, oBits);
			Synchronize();
		}

		delete [] andResLeft;
		counter++;
	}

	for(int i=0; i < oBits; i ++) {
                Copy(ctRes[i], tempSum[i]);
        }
	Synchronize();
	for (int i = 0; i < kNumSMs; i ++)
		st[i].Destroy();
	delete [] st;
	delete [] tempSum;
	delete [] andRes;
}


// function to convert decimal to binary
std::string decToBinary(int n)
{
    std::string r;
    while(n!=0) {r=(n%2==0 ?"0":"1")+r; n/=2;}
    return r;
}

// function to convert binary to decimal
std::string binToDecimal(string s)
{
    unsigned long long value = std::stoull(s, 0, 2);
    std::string str = std::to_string(value);
    return str;
}


string add(string a, string b){
   string result = "";
   int temp = 0;
   int size_a = a.size() - 1;
   int size_b = b.size() - 1;
   while (size_a >= 0 || size_b >= 0 || temp == 1){
      temp += ((size_a >= 0)? a[size_a] - '0': 0);
      temp += ((size_b >= 0)? b[size_b] - '0': 0);
      result = char(temp % 2 + '0') + result;
      temp /= 2;
      size_a--; size_b--;
   }
   return result;
}

// function to convert to Two's Complement
std::string toTwoComplement(string s) {
  
  int size = s.length();
  
  char binary[size + 1], one[size + 1], two[size + 1];
  int i, carry = 1, fail = 0;

  strcpy(binary, s.c_str());

  for (i = 0; i < size; i++) {
    if (binary[i] == '1') {
      one[i] = '0';
    } else if (binary[i] == '0') {
      one[i] = '1';
    }
  }
  one[size] = '\0';

  return add(one,"1");
}


// function to add missing zeros
std::string addZeros(string b, int bits){
   for ( int i = b.length(); i < bits; i++ ){
       b = "0" + b;
   };
   return b;
};


class Server_socket{

    fstream file;
    fstream file2;

    int PORT;
    int check;

    int general_socket_descriptor;
    int new_socket_descriptor;
    struct sockaddr_in address;
    int address_length;

    public:
        Server_socket(){

	};

	void start_everything(int number){
            create_socket();

            PORT = number;

            cout << " The port is: " << PORT << "\n";
	    address.sin_family = AF_INET;
            address.sin_addr.s_addr = INADDR_ANY;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

            bind_socket();
            set_listen_set();
            accept_connection();

            file.open("cipherRes/overall", ios::in | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }
        }

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) == 0) {
                perror("[ERROR] : Socket failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
        }

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))!=0) {

                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void accept_connection(){
            if ((new_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void transmit_file(){
            std::string contents((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(new_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";
	    cout<<"===============================\n";
        }

	void close_socket(){
            close(new_socket_descriptor);
	}

	void split_file(int numBits){
            int count = 0;
	    std::ifstream file("cipher/overall");

	    //last one will be for publickey
	    std::string filenames[numBits * 2 + 1];
            for (int i = 0; i < numBits * 2; i ++){
                string filename = "cipher/ct" + std::to_string(i);
		remove(filename.c_str());
		filenames[i] = filename;
	    };

	    filenames[numBits * 2]="finalkeys/publickey1.txt";
	    remove("finalkeys/publickey1.txt");


	    if (file.is_open()) {
    	 	std::string line;
    		while (std::getline(file, line)) {

		      if(count==(501*(numBits*2))){
	    	      	ofstream pubkey; 
                      	pubkey.open("finalkeys/publickey1.txt",fstream::app);
	              	pubkey << line.c_str() << endl;

		      } else {
		      	int fileChoice = floor(count/501);
	              	ofstream Myfile;
		      	Myfile.open(filenames[fileChoice], fstream::app);
		      	Myfile << line.c_str() << endl;
		      	count += 1;

		      }

	        };
	     };
        };

	void receive_file(){

            file2.open("cipher/overall", ios::out | ios::trunc | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Return File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

	    char buffer[2200024] = {};
	    bzero(buffer, sizeof(buffer));
	    int count = 0;
	    printf("Starting to download file contents");
	    while(1){
                    printf("Beginning file contents");
	            int valread = read(new_socket_descriptor , buffer, 2200024);
		    printf("%d",valread);
		    if(valread == 0)
			    break;
		    printf("%s", buffer);
		    file2<<buffer;
		    bzero(buffer, sizeof(buffer));
	    };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
	    file2.close();

	    };


};

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  uint32_t kNumLevels = 4;
  int numBits = 256;
  int port1 = 4380;
  int port2 = 4381;

  SetSeed(); // set random seed


  Ptxt* pt = new Ptxt[numBits];
  Ptxt* pt1 = new Ptxt[numBits];
  Ptxt* ptRes = new Ptxt[numBits];
  Ctxt* ct = new Ctxt[numBits];
  Ctxt* ct1 = new Ctxt[numBits];
  Ctxt* ctRes = new Ctxt[numBits];
  Ctxt *minusEnd = new Ctxt[numBits];

  Synchronize();
  bool correct;
  correct = true;

  PubKey pub_key;
  PriKey pri_key;
  KeyGen(pub_key,pri_key);

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing

/*
  for (int i = 0; i<numBits; i++){
	  pt[i].message_ = rand() % Ptxt::kPtxtSpace;
	  pt1[i].message_ = rand() % Ptxt::kPtxtSpace;
  }
*/



  // Getting the User Inputs ========================
   unsigned long long input1, input2, operator_code, bits;
   string sign; 
   bool x_neg = false;
   bool y_neg = false;
  
   // Get inputs
   cout << "How many bits do you want: ";
   cin >> bits;
   cout << "What is your first number: ";
   cin >> input1;
   cout << "What is your second number: ";
   cin >> input2;
   cout << "What is your operator: ";
   cin >> sign;

   //cout << "Your equation is: " << input1 << sign << input2 << "\n";

   // Check Negative 
   if (input1 < 0){
       x_neg = true;
   };
   if (input2 < 0){
       y_neg = true;
   };
   
   // Check Addition
   if (sign == "+"){
       if (!x_neg && !y_neg){
	   operator_code = 0;
       } else if (!x_neg && y_neg){
	   operator_code = 1;
       } else if (x_neg && !y_neg){
	   operator_code = 2;
       } else {
	   operator_code = 3;
       };
   };

   // Check Subtraction
   if (sign == "-"){
       if (!x_neg && !y_neg){
	   operator_code = 1;
       } else if (!x_neg && y_neg){
	   operator_code = 0;
       } else if (x_neg && !y_neg){
	   operator_code = 3;
       } else {
	   operator_code = 2;
       };
   };

   // Check Division
   if (sign == "*"){
       if (!x_neg && !y_neg){
	   operator_code = 4;
       } else if (!x_neg && y_neg){
	   operator_code = 5;
       } else if (x_neg && !y_neg){
	   operator_code = 5;
       } else {
	   operator_code = 4;
       };
   };

   
   // Conver Decimal to Binary
   string x, y;

   x = decToBinary(input1);
   y = decToBinary(input2);

   // Add the missing zeros
   x = addZeros(x ,bits);
   y = addZeros(y, bits);

   for ( int i = 0; i < numBits; i++){
       pt[i] = x[i];
       pt1[i] = y[i];
   }
   cout << "\nThe operator code is: " << operator_code << "\n";
   // Write Operator Code 
   ofstream myfile;
   myfile.open ("operator.txt");
   myfile << operator_code;
   myfile.close();

  // ================================================================


  for (int i = 0; i < numBits; i ++) {
        Encrypt(ct[i], pt[i], pri_key);
        Encrypt(ct1[i], pt1[i], pri_key);
  }

/*
  for (int i = 0; i < numBits; i ++) {
	  ReadCtxtFromFile(ct[i],"cipher/ct"+std::to_string(i));
  }

  int countCT=0;
  for (int i = 32; i < numBits + numBits; i ++) {
	  ReadCtxtFromFile(ct1[countCT],"cipher/ct"+std::to_string(i));
	  countCT++;
  }

*/

  //cout<< "Number of tests:\t" << numBits <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();


  Synchronize();


  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  Ctxt* zero = new Ctxt[numBits];
  Ctxt* temp = new Ctxt[numBits];
  And(zero[numBits - 1],ct[0],ct1[0],st[0 % kNumSMs]);
  Xor(zero[numBits - 1],zero[0],zero[0],st[0 % kNumSMs]);
  Synchronize();

  Ctxt* one = new Ctxt[1];
  Not(one[0], zero[numBits - 1]);

  for ( int i = 0; i < numBits - 1 ; i++ ){
    Copy(zero[i], one[0]);
  };

  if (operator_code == 0 ){
      addNumbers(ctRes, ct, ct1, numBits);
  } else if ( operator_code == 1){
      subNumbers(ctRes, ct, ct1, numBits);
  } else if ( operator_code == 2){
      subNumbers(ctRes, ct1, ct, numBits);
  } else if ( operator_code == 3){ 
      addNumbers(ctRes, ct, ct1, numBits);

      for ( int i = 0; i < numBits; i++ ){
	    Not(temp[i], ctRes[i]);
      };
      Synchronize();

      addNumbers(ctRes, zero, temp, numBits);
      
      Not(ctRes[0], ctRes[0]);

  } else if ( operator_code == 4){
      mulNumbers(ctRes, ct, ct1, (numBits/2), numBits);
  } else if ( operator_code == 5){
      mulNumbers(ctRes, ct, ct1, (numBits/2), numBits);
     
      for ( int i = 0; i < numBits; i++ ){
             Not(temp[i], ctRes[i]);
      };    
      Synchronize();

      addNumbers(ctRes, zero, temp, numBits);
 
      Not(ctRes[0], ctRes[0]);
  };



  // Here, pass streams to gates for parallel gates.
  // addNumbers(ctRes, ct,ct1,32);
  // mulNumbers(ctRes, ct,ct1,16,32);
  // subNumbers(ctRes, ct, ct1, 32);

  Synchronize();


  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&et, start, stop);
  cout<< et / kNumLevels << " ms for addition" <<endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);
/*
    cout << "\nINPUT ONE:\n";
    for (int i=0; i < numBits; i++) {
            cout << pt[i].message_;
    }


    cout << "\nINPUT TWO:\n";
    for (int i=0; i < numBits; i++) {
            cout << pt1[i].message_;
    }
*/
    //subNumbers(ctRes, ct, ct1, 32);

  
    for (int i=0; i < numBits; i++) {
        Decrypt(ptRes[i], ctRes[i], pri_key);
    }
    
    std::string result;

    cout << "\nRESULT:\n";
    for (int i=0; i < numBits; i++) {
            //cout << ptRes[i].message_;
	    result = result + std::to_string(ptRes[i].message_);
    }
    cout << "\n The result is : " << result;
    if(result[0] == '1'){
	result = toTwoComplement(result);
	result = "-" + binToDecimal(result);
    } else {
        result = binToDecimal(result);
    };

    cout << "\n The result is: " << result;

   


  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  
  delete [] st;

  cout<< "\n------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  return 0;
}
