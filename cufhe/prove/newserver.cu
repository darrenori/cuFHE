#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include<iostream>
#include<fstream>
#include<stdio.h>
#include <unistd.h>
#include <sys/socket.h>
#include <stdlib.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <strings.h>
#include <string>
#include <cmath>



#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

void NotCheck(Ptxt& out, const Ptxt& in) {
	out.message_ = (~in.message_) & 0x1;
}

void CopyCheck(Ptxt& out, const Ptxt& in){
	out.message_ = in.message_;
}

void addBits(Ctxt *r, Ctxt &a, Ctxt &b, Ctxt *carry) {
	Ctxt *t1 = new Ctxt[1];
    Ctxt *t2 = new Ctxt[1];
	Xor(t1[0], a, carry[0]);
    Xor(t2[0], b, carry[0]);
	Synchronize();
	Xor(r[0], a, t2[0]);
	And(t1[0], t1[0], t2[0]);
	Synchronize();
	Xor(r[1], carry[0], t1[0]);
	Synchronize();
	delete [] t1;
	delete [] t2;
}

void addNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
	
	Ctxt *carry = new Ctxt[1];
    Ctxt *bitResult = new Ctxt[2];

	Xor(ctRes[0], ctA[0], ctB[0]);
	And(carry[0], ctA[0], ctB[0]);
	Synchronize();
	for(int i = 1; i < nBits; i++) {
		addBits(bitResult, ctA[i], ctB[i], carry);
		Copy(ctRes[i], bitResult[0]);
		Copy(carry[0], bitResult[1]);
		Synchronize();
	}
	delete [] carry;
	delete [] bitResult;
}


class Client_socket{
    fstream file;

    int PORT;
    
    int general_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:
        Client_socket(){
	
	};

	void start_everything(int number){
            create_socket();
            PORT = number;

	    cout << "The port is " << PORT << "\n";

            address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);
            if(inet_pton(AF_INET, "69.69.69.1", &address.sin_addr)<=0) { 
                cout<<"[ERROR] : Invalid address\n";
            }

            create_connection();
            
            file.open("rec.txt", ios::out | ios::trunc | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }
        };

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) < 0) { 
                perror("[ERROR] : Socket failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
        }

        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) { 
                perror("[ERROR] : connection attempt failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connection Successfull.\n";
        }

        void close_socket(){
	    close(general_socket_descriptor);
	};

        void receive_file(){
            char buffer[2200024] = {};
	    bzero(buffer, sizeof(buffer));
	    int count = 0;
	    while(1){

	            int valread = read(general_socket_descriptor , buffer, 2200024);
		    if(valread == 0)
			    break;
		    file<<buffer;
		    bzero(buffer, sizeof(buffer));
	    };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
	    file.close();
	    close(general_socket_descriptor);
        }

	void split_file(){
            int count = 0;
	    std::ifstream file("rec.txt");
	    
	    std::string filenames[64];
            for (int i = 0; i < 64; i ++){
                string filename = "cipher/Ctxt" + std::to_string(i);
		filenames[i] = filename;
	    };

	    if (file.is_open()) {
    	 	std::string line;
    		while (std::getline(file, line)) {
		      int fileChoice = floor(count/501);
	              ofstream Myfile;
		      Myfile.open(filenames[fileChoice], fstream::app);
		      Myfile << line.c_str() << endl;
		      count += 1; 
	        };
	     };
        };
 
        void transmit_file(){

	    fstream file2;
            
            printf("============================\n");	    
	    file2.open("cipherRes/overall", ios::in | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Send File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            	    
            std::string contents((std::istreambuf_iterator<char>(file2)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";	
	}

}





int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
 // uint32_t kNumTests = kNumSMs * 32;// * 8;
  uint32_t kNumLevels = 4;
 // uint32_t val1 = 1;
 // uint32_t val2 = 2;
  int numBits = 32;
  int port1 = 4380;
  int port2 = 4381;

  //SetSeed(); // set random seed

  PriKey pri_key; // public key
  PubKey pub_key;

  ReadPubKeyFromFile(pub_key,"finalkeys/publickey1.txt");
  remove("cipher/overall");

  Ptxt* pt = new Ptxt[numBits];
  Ptxt* pt1 = new Ptxt[numBits];
  Ptxt* ptRes = new Ptxt[numBits];
  Ctxt* ct = new Ctxt[numBits];
  Ctxt* ct1 = new Ctxt[numBits];
  Ctxt* ctRes = new Ctxt[numBits];
  Synchronize();
  bool correct;
  correct = true;

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing



  //RECEIVE DATA FROM CLIENT!
  Client_socket C;
  c.start_everything(port1);
  C.receive_file();
  C.split_file();
  C.close_socket();
  return 0;


  for (int i = 0; i < numBits; i ++) {
	  ReadCtxtFromFile(ct[i],"cipher/ct"+std::to_string(i));
  }

  for (int i = 32; i < numBits; i ++) {
	  ReadCtxtFromFile(ct1[i],"cipher/ct"+std::to_string(i));
  }


  cout<< "Number of tests:\t" << numBits <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();


  Synchronize();





  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  // Here, pass streams to gates for parallel gates.
  cout<< "------ Test NAND Gate ------" <<endl;
  for (int i = 0; i < numBits; i ++) {
    Nand(ctRes[i], ct[i], ct1[i], st[i % kNumSMs]);
  }

  Synchronize();

  for (int i = 0; i < numBits; i ++) {
	  WriteCtxtToFile(ctRes[i],"cipherRes/ct"+std::to_string(i));
  }
  
  //only 32 files for cipheres
  for (int i = 0; i < numBits; i ++) {
            std::ifstream if_a("cipherRes/ct"+std::to_string(i),std::ios_base::app);
            std::ofstream of_c("cipherRes/overall",std::ios_base::app);
            of_c << if_a.rdbuf();
  }


  

  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  
  delete [] st;

  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  return 0;
}
