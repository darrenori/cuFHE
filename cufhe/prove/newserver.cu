/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include<iostream>
#include<fstream>
#include<stdio.h>
#include <unistd.h>
#include <sys/socket.h>
#include <stdlib.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <strings.h>
#include <string>
#include <cmath>



#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

void NotCheck(Ptxt& out, const Ptxt& in) {
	out.message_ = (~in.message_) & 0x1;
}

void CopyCheck(Ptxt& out, const Ptxt& in){
	out.message_ = in.message_;
}

void addBits(Ctxt *r, Ctxt &a, Ctxt &b, Ctxt *carry) {
	Ctxt *t1 = new Ctxt[1];
    Ctxt *t2 = new Ctxt[1];
	Xor(t1[0], a, carry[0]);
    Xor(t2[0], b, carry[0]);
	Synchronize();
	Xor(r[0], a, t2[0]);
	And(t1[0], t1[0], t2[0]);
	Synchronize();
	Xor(r[1], carry[0], t1[0]);
	Synchronize();
	delete [] t1;
	delete [] t2;
}

void addNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
	
	Ctxt *carry = new Ctxt[1];
    Ctxt *bitResult = new Ctxt[2];

	Xor(ctRes[0], ctA[0], ctB[0]);
	And(carry[0], ctA[0], ctB[0]);
	Synchronize();
	for(int i = 1; i < nBits; i++) {
		addBits(bitResult, ctA[i], ctB[i], carry);
		Copy(ctRes[i], bitResult[0]);
		Copy(carry[0], bitResult[1]);
		Synchronize();
	}
	delete [] carry;
	delete [] bitResult;
}





int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
 // uint32_t kNumTests = kNumSMs * 32;// * 8;
  uint32_t kNumLevels = 4;
 // uint32_t val1 = 1;
 // uint32_t val2 = 2;
  int numBits = 32;
  int port1 = 4380;
  int port2 = 4381;

  //SetSeed(); // set random seed

  PriKey pri_key; // public key
  PubKey pub_key;

  ReadPubKeyFromFile(pub_key,"finalkeys/publickey1.txt");
  remove("cipher/overall");

  Ptxt* pt = new Ptxt[numBits];
  Ptxt* pt1 = new Ptxt[numBits];
  Ptxt* ptRes = new Ptxt[numBits];
  Ctxt* ct = new Ctxt[numBits];
  Ctxt* ct1 = new Ctxt[numBits];
  Ctxt* ctRes = new Ctxt[numBits];
  Synchronize();
  bool correct;
  correct = true;

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing



  //RECEIVE DATA FROM CLIENT!
  Server_socket S;
  S.start_everything(port1);
  S.transmit_file();
  S.close_socket();
  return 0;


  for (int i = 0; i < numBits; i ++) {
	  ReadCtxtFromFile(ct[i],"cipher/ct"+std::to_string(i));
  }

  for (int i = 32; i < numBits; i ++) {
	  ReadCtxtFromFile(ct1[i],"cipher/ct"+std::to_string(i));
  }


  cout<< "Number of tests:\t" << numBits <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();


  Synchronize();





  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  // Here, pass streams to gates for parallel gates.
  cout<< "------ Test NAND Gate ------" <<endl;
  for (int i = 0; i < numBits; i ++) {
    Nand(ctRes[i], ct[i], ct1[i], st[i % kNumSMs]);
  }

  Synchronize();

  for (int i = 0; i < numBits; i ++) {
	  WriteCtxtToFile(ctRes[i],"cipherRes/ct"+std::to_string(i));
  }
  
  //only 32 files for cipheres
  for (int i = 0; i < numBits; i ++) {
            std::ifstream if_a("cipherRes/ct"+std::to_string(i),std::ios_base::app);
            std::ofstream of_c("cipherRes/overall",std::ios_base::app);
            of_c << if_a.rdbuf();
  }


  

  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  
  delete [] st;

  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  return 0;
}
