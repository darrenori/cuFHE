#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include<iostream>
#include<fstream>
#include<stdio.h>
#include <unistd.h>
#include <sys/socket.h>
#include <stdlib.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <strings.h>
#include <string>
#include <cmath>



#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

void NotCheck(Ptxt& out, const Ptxt& in) {
	out.message_ = (~in.message_) & 0x1;
}

void CopyCheck(Ptxt& out, const Ptxt& in){
	out.message_ = in.message_;
}


void addBits(Ctxt *r, Ctxt &a, Ctxt &b, Ctxt *carry) {
        Ctxt *t1 = new Ctxt[1];
    Ctxt *t2 = new Ctxt[1];
        Xor(t1[0], a, carry[0]);
    Xor(t2[0], b, carry[0]);
        Synchronize();
        Xor(r[0], a, t2[0]);
        And(t1[0], t1[0], t2[0]);
        Synchronize();
        Xor(r[1], carry[0], t1[0]);
        Synchronize();
        delete [] t1;
        delete [] t2;
}

void addNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {


        Ctxt *carry = new Ctxt[1];
    Ctxt *bitResult = new Ctxt[2];

        Xor(ctRes[31], ctA[31], ctB[31]);
        And(carry[0], ctA[31], ctB[31]);
        Synchronize();
        for(int i = 30; i > 0; i--) {
                addBits(bitResult, ctA[i], ctB[i], carry);
                Copy(ctRes[i], bitResult[0]);
                Copy(carry[0], bitResult[1]);
                Synchronize();
        }
        Copy(ctRes[0], carry[0]);
        //Copy(ctRes[nBits-1],carry[0]);

        Synchronize();
        delete [] carry;
        delete [] bitResult;
}




class Server_socket{

    fstream file;
    fstream file2;

    int PORT;
    int check;

    int general_socket_descriptor;
    int new_socket_descriptor;
    struct sockaddr_in address;
    int address_length;

    public:
        Server_socket(){

	};

	void start_everything(int number){
            create_socket();

            PORT = number;

            cout << " The port is: " << PORT << "\n";
	    address.sin_family = AF_INET;
            address.sin_addr.s_addr = INADDR_ANY;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

            bind_socket();
            set_listen_set();
            accept_connection();

            file.open("cipherRes/overall", ios::in | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }
        }

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) == 0) {
                perror("[ERROR] : Socket failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
        }

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))!=0) {

                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void accept_connection(){
            if ((new_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void transmit_file(){
            std::string contents((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(new_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";
	    cout<<"===============================\n";
        }

	void close_socket(){
            close(new_socket_descriptor);
	}

	void split_file(){
            int count = 0;
	    std::ifstream file("cipher/overall");

	    //last one will be for publickey
	    std::string filenames[65];
            for (int i = 0; i < 64; i ++){
                string filename = "cipher/ct" + std::to_string(i);
		remove(filename.c_str());
		filenames[i] = filename;
	    };

	    filenames[64]="finalkeys/publickey1.txt";
	    remove("finalkeys/publickey1.txt");


	    if (file.is_open()) {
    	 	std::string line;
    		while (std::getline(file, line)) {

		      if(count==(501*64)){
	    	      	ofstream pubkey;
                      	pubkey.open("finalkeys/publickey1.txt",fstream::app);
	              	pubkey << line.c_str() << endl;

		      } else {
		      	int fileChoice = floor(count/501);
	              	ofstream Myfile;
		      	Myfile.open(filenames[fileChoice], fstream::app);
		      	Myfile << line.c_str() << endl;
		      	count += 1;

		      }

	        };
	     };
        };

	void receive_file(){

            file2.open("cipher/overall", ios::out | ios::trunc | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Return File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

	    char buffer[2200024] = {};
	    bzero(buffer, sizeof(buffer));
	    int count = 0;
	    printf("Starting to download file contents");
	    while(1){
                    printf("Beginning file contents");
	            int valread = read(new_socket_descriptor , buffer, 2200024);
		    printf("%d",valread);
		    if(valread == 0)
			    break;
		    printf("%s", buffer);
		    file2<<buffer;
		    bzero(buffer, sizeof(buffer));
	    };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
	    file2.close();

	    };


};

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
 // uint32_t kNumTests = kNumSMs * 32;// * 8;
  uint32_t kNumLevels = 4;
 // uint32_t val1 = 1;
 // uint32_t val2 = 2;
  int numBits = 32;
  int port1 = 4380;
  int port2 = 4381;

  //SetSeed(); // set random seed


  remove("cipher/overall");

  Ptxt* pt = new Ptxt[numBits];
  Ptxt* pt1 = new Ptxt[numBits];
  Ptxt* ptRes = new Ptxt[numBits];
  Ctxt* ct = new Ctxt[numBits];
  Ctxt* ct1 = new Ctxt[numBits];
  Ctxt* ctRes = new Ctxt[numBits];
  Synchronize();
  bool correct;
  correct = true;

  //RECEIVE DATA FROM CLIENT!
  Server_socket S;
  S.start_everything(port1);
  S.receive_file();
  S.split_file();



  PubKey pub_key;
  ReadPubKeyFromFile(pub_key,"finalkeys/publickey1.txt");

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing





  for (int i = 0; i < numBits; i ++) {
	  ReadCtxtFromFile(ct[i],"cipher/ct"+std::to_string(i));
  }

  int countCT=0;
  for (int i = 32; i < numBits + numBits; i ++) {
	  ReadCtxtFromFile(ct1[countCT],"cipher/ct"+std::to_string(i));
	  countCT++;
  }


  cout<< "Number of tests:\t" << numBits <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();


  Synchronize();





  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  // Here, pass streams to gates for parallel gates.
  addNumbers(ctRes, ct,ct1,32);
  /*
  cout<< "------ Test NAND Gate ------" <<endl;
  for (int i = 0; i < numBits; i ++) {
    Nand(ctRes[i], ct[i], ct1[i], st[i % kNumSMs]);
  }
*/
  Synchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&et, start, stop);
  cout<< et / kNumLevels << " ms for addition" <<endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);


  string fname;
  remove("cipherRes/overall");
  for (int i = 0; i < numBits; i ++) {
	  fname = "cipherRes/ct"+std::to_string(i);
	  remove(fname.c_str());
	  WriteCtxtToFile(ctRes[i],fname);
  }
  
  //only 32 files for cipheres
  for (int i = 0; i < numBits; i ++) {
            std::ifstream if_a("cipherRes/ct"+std::to_string(i),std::ios_base::app);
            std::ofstream of_c("cipherRes/overall",std::ios_base::app);
            of_c << if_a.rdbuf();
  }

  Server_socket S1;
  S1.start_everything(port2);
  S1.transmit_file();

  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  
  delete [] st;

  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  return 0;
}
