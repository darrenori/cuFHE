#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
#include <netinet/in.h>

#include <ios>

  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;



class Client_socket{
    fstream file;

    int PORT;
    
    int general_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:
        Client_socket(){
	
	};

	void start_everything(int number){
            create_socket();
            PORT = number;

	    cout << "The port is " << PORT << "\n";

            address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);
            if(inet_pton(AF_INET, "127.0.0.1", &address.sin_addr)<=0) { 
                cout<<"[ERROR] : Invalid address\n";
            }

            create_connection();
            
            file.open("rec.txt", ios::out | ios::trunc | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }
        };

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) < 0) { 
                perror("[ERROR] : Socket failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
        }

        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) { 
                //perror("[ERROR] : connection attempt failed.\n");
		sleep(10);
            	create_connection();
                //exit(EXIT_FAILURE);
            } else {
            	cout<<"[LOG] : Connection Successfull.\n";
            }
        }

        void close_socket(){
	    close(general_socket_descriptor);
	};

        void receive_file(){
            char buffer[2200024] = {};
	    bzero(buffer, sizeof(buffer));
	    int count = 0;
	    while(1){

	            int valread = read(general_socket_descriptor , buffer, 2200024);
		    if(valread == 0)
			    break;
		    file<<buffer;
		    bzero(buffer, sizeof(buffer));
	    };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
	    file.close();
	    close(general_socket_descriptor);
        }

	void split_file(){
            int count = 0;
	    std::ifstream file("rec.txt");
	    
	    std::string filenames[32];
            for (int i = 0; i < 32; i ++){
                string filename = "cipherRes/ct" + std::to_string(i);
		remove(filename.c_str());
		filenames[i] = filename;
	    };

	    if (file.is_open()) {
    	 	std::string line;
    		while (std::getline(file, line)) {
		      int fileChoice = floor(count/501);
	              ofstream Myfile;
		      Myfile.open(filenames[fileChoice], fstream::app);
		      Myfile << line.c_str() << endl;
		      count += 1; 
	        };
	     };
        };
 
        void transmit_file(){

	    fstream file2;
            
            printf("============================\n");	    
	    file2.open("cipher/overall", ios::in | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Send File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            	    
            std::string contents((std::istreambuf_iterator<char>(file2)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";	
	}

};


void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}


// function to convert decimal to binary
std::string decToBinary(int n)
{
    std::string r;
    while(n!=0) {r=(n%2==0 ?"0":"1")+r; n/=2;}
    return r;
}

// function to convert binary to decimal
std::string binToDecimal(string s)
{
    unsigned long long value = std::stoull(s, 0, 2);
    std::string str = std::to_string(value);
    return str;
}

// function to convert to Two's Complement
std::string toTwoComplement(string s) {

  int size = s.length();

  char binary[size + 1], one[size + 1], two[size + 1];
  int i, carry = 1, fail = 0;

  strcpy(binary, s.c_str());

  for (i = 0; i < size; i++) {
    if (binary[i] == '1') {
      one[i] = '0';
    } else if (binary[i] == '0') {
      one[i] = '1';
    }
  }
  one[size] = '\0';

  return add(one,"1");
}


// function to add missing zeros
std::string addZeros(string b, int bits){
   for ( int i = b.length(); i < bits; i++ ){
       b = "0" + b;
   };
   return b;
};

int main(int argc, char const* argv[])
{
    int port1 = 4380;
    int port2 = 4381;
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    SetSeed();


    PriKey pri_key; // private key
    PubKey pub_key;
    bool correct = true;

    struct timeval start, end;
    double get_time;
    gettimeofday(&start, NULL);

    KeyGen(pub_key, pri_key);

    gettimeofday(&end, NULL);
    get_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1.0E-6;
    printf("Computation Time: %lf[sec]\n", get_time);


    WritePubKeyToFile(pub_key,"finalkeys/publickey1.txt");
    Initialize(pub_key);

    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];



    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();

    / Getting the User Inputs ========================
   int input1, input2, operator_code, bits;
   string sign;
   bool x_neg = false;
   bool y_neg = false;

   // Get inputs
   cout << "How many bits do you want: ";
   cin >> bits;
   cout << "What is your first number: ";
   cin >> input1;
   cout << "What is your second number: ";
   cin >> input2;
   cout << "What is your operator: ";
   cin >> sign;

   //cout << "Your equation is: " << input1 << sign << input2 << "\n";

   // Check Negative
   if (input1 < 0){
       x_neg = true;
   };
   if (input2 < 0){
       y_neg = true;
   };

   // Check Addition
   if (sign == "+"){
       if (!x_neg && !y_neg){
	   operator_code = 0;
       } else if (!x_neg && y_neg){
	   operator_code = 1;
       } else if (x_neg && !y_neg){
	   operator_code = 2;
       } else {
	   operator_code = 3;
       };
   };

   // Check Subtraction
   if (sign == "-"){
       if (!x_neg && !y_neg){
	   operator_code = 1;
       } else if (!x_neg && y_neg){
	   operator_code = 0;
       } else if (x_neg && !y_neg){
	   operator_code = 3;
       } else {
	   operator_code = 2;
       };
   };

   // Check Division
   if (sign == "*"){
       if (!x_neg && !y_neg){
	   operator_code = 4;
       } else if (!x_neg && y_neg){
	   operator_code = 5;
       } else if (x_neg && !y_neg){
	   operator_code = 5;
       } else {
	   operator_code = 4;
       };
   };


   // Conver Decimal to Binary
   string x, y;

   x = decToBinary(input1);
   y = decToBinary(input2);

   // Add the missing zeros
   x = addZeros(x ,bits);
   y = addZeros(y, bits);

   for ( int i = 0; i < 32; i++){
       pt[i] = x[i];
       pt1[i] = y[i];
   }

   cout << "\nThe operator code is: " << operator_code << "\n";
   // Write Operator Code
   ofstream myfile;
   myfile.open ("operator.txt");
   myfile << operator_code << "\n";
   myfile.close();
/*

    for (int i = 0; i < numBits; i ++) {
	    pt[i] = rand() % Ptxt::kPtxtSpace;
	    pt1[i] = rand() % Ptxt::kPtxtSpace;
    }
*/


    //FOR DEBUGGING
/*
    cout << "INPUT ONE\n";
    for (int i = 0; i < numBits; i ++) {
	    cout << pt[i].message_;
    }

    cout<< "\nINPUT TWO\n";
    for (int i = 0; i < numBits; i ++) {
	    cout << pt1[i].message_;
    }

*/
    for (int i = 0; i < numBits; i ++) {
   	Encrypt(ct[i], pt[i], pri_key);
      	Encrypt(ct1[i], pt1[i], pri_key);
    }


    Synchronize();





    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
	    filename = "cipher1/ct" + std::to_string(i);
	    WriteCtxtToFile(ct1[i],filename);
    }

    remove("cipher/overall");

    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }

    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher1/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }

    //transmit public keys
    std::ifstream if_a("finalkeys/publickey1.txt",std::ios_base::app);
    std::ofstream of_c("cipher/overall",std::ios_base::app);
    of_c << if_a.rdbuf();

    
    gettimeofday(&start, NULL);

    Client_socket C;
    C.start_everything(port1);
    C.transmit_file();
    C.close_socket();

    gettimeofday(&end, NULL);
    get_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1.0E-6;
    printf("First send: %lf[sec]\n", get_time);



    gettimeofday(&start, NULL);

   Client_socket C1;
   C1.start_everything(port2);
   C1.receive_file();
   C1.split_file();



    //-------------------READING BACK DATA FROM SERVER----------------------//
    for (int i = 0; i < numBits; i ++) {
            string filename = "cipherRes/ct" + std::to_string(i);
            ReadCtxtFromFile(ctRes[i],filename);
    }


    cout << "\nINPUT ONE:\n";
    for (int i=0; i < numBits; i++) {
	    cout << pt[i].message_;
    }


    cout << "\nINPUT TWO:\n";
    for (int i=0; i < numBits; i++) {
	    cout << pt1[i].message_;
    }


/*

    for (int i=0; i < numBits; i++) {
	Decrypt(pt1[i], ctRes[i], pri_key);
    }



    cout << "\nRESULT:\n";
    for (int i=0; i < numBits; i++) {
	    cout << pt1[i].message_;
    }
 */

    //READ COMPUTED DATA FROM SERVER HERE!
    int cnt_failures = 0;
    for (int i = 0; i < numBits; i ++) {
      Decrypt(ptRes[i], ctRes[i], pri_key);
      /*if (pt1[i].message_ != ptRes[i].message_) {
        std::cout << "FAILED" << pt1[i].message_ << "||" <<ptRes[i].message_ << "\n";
        correct = false;
        cnt_failures += 1;
        //std::cout<< "Fail at iteration: " << i <<std::endl;
      }*/
    }

    

    //for debugging ONLY
/*
    cout << "PLAINTEXT RESULT\n";
    for(int i=0; i< numBits; i++){
	    cout << ptRes[i].message_;
    }



*/

     std::string result;

    cout << "\nRESULT:\n";
    for (int i=0; i < numBits; i++) {
            //cout << ptRes[i].message_;
	    result = result + std::to_string(ptRes[i].message_);
    }
    cout << "\n The result is : " << result;
    if(result[0] == '1'){
	result = toTwoComplement(result);
	result = "-" + binToDecimal(result);
    } else {
        result = binToDecimal(result);
    };

    cout << "\n The result is: " << result;

    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;


}
