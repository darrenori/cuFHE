#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
#include <netinet/in.h>

#include <ios>

  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>

using namespace std;

class Client_socket{
    fstream file;

    int PORT;
    
    int general_socket_descriptor;
    int new_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:

            void start_everything(int number, string role){
            create_socket();
            PORT = number;

            cout << "The port is " << PORT << "\n";

            address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

            if ( role == "server" ){
                
	        address.sin_addr.s_addr = INADDR_ANY;
                bind_socket();
                set_listen_set();
                accept_connection();

            } else {

                if(inet_pton(AF_INET, "192.168.0.1", &address.sin_addr)<=0) {
                        cout<<"[ERROR] : Invalid address\n";
                }

                create_connection();

                file.open("rec.txt", ios::out | ios::trunc | ios::binary);
                if(file.is_open()){
                        cout<<"[LOG] : File Created.\n";
                }
                else{
                        cout<<"[ERROR] : File creation failed, Exititng.\n";
                        exit(EXIT_FAILURE);
                }
            }
        };

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))!=0) {

                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void accept_connection(){
            if ((general_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) < 0) { 
                perror("[ERROR] : Socket failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
	    const int enable = 1;
            if (setsockopt(general_socket_descriptor, SOL_SOCKET, SO_REUSEADDR, &enable, sizeof(int)) < 0)
                   perror("setsockopt(SO_REUSEADDR) failed");

        }

        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) { 
                //perror("[ERROR] : connection attempt failed.\n");
		sleep(10);
            	create_connection();
                //exit(EXIT_FAILURE);
            } else {
            	cout<<"[LOG] : Connection Successfull.\n";
            }
        }

        void close_socket(){
	    close(general_socket_descriptor);
	};

 
        void transmit_file(){

	    fstream file2;
            
            printf("============================\n");	    
	    file2.open("cipher/overall", ios::in | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Send File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            	    
            std::string contents((std::istreambuf_iterator<char>(file2)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";	
	}

	    void receive_file(){

            remove("finalkeys/privatekey1.txt");

            fstream file2;

            file2.open("finalkeys/privatekey1.txt", ios::out | ios::trunc | ios::binary);
            	if(file2.is_open()){
                	cout<<"[LOG] : Return File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            char buffer[2200024] = {};
            bzero(buffer, sizeof(buffer));
            printf("Starting to download file contents");
            while(1){
                    printf("Beginning file contents");
                    int valread = read(general_socket_descriptor , buffer, 2200024);
                    printf("%d",valread);
                    if(valread == 0)
                            break;
                    printf("%s", buffer);
                    file2<<buffer;
                    bzero(buffer, sizeof(buffer));
            };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
            file2.close();

            };
};


// function to convert decimal to binary
std::string decToBinary(int n)
{
    std::string r;
    while(n!=0) {r=(n%2==0 ?"0":"1")+r; n/=2;}
    return r;
}

// function to convert binary to decimal
std::string binToDecimal(string s)
{
    unsigned long long value = std::stoull(s, 0, 2);
    std::string str = std::to_string(value);
    return str;
}

std::string addZeros(string b, int bits){
    for ( int i = b.length(); i < bits; i++ ){
	 b = "0" + b;
    }
    return b;
};

int main(int argc, char const* argv[])
{

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    int numBits = 32;

    SetSeed();

    // Get Private Key from Key Generation
    Client_socket K;
    K.start_everything(4384, "server");
    K.receive_file();
    K.close_socket();

    PriKey pri_key; // private key

    struct timeval start, end;
    double get_time;


    ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();

   // Getting the User Inputs ========================
   int input1, operator_code;
   string sign;

   // Get inputs
   cout << "How many bits do you want: ";
   cin >> numBits;
   
   cout << "What is your first number: ";
   cin >> input1;

   
    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];
   if ( input1 < 0 ){
	   operator_code = 2;
   } else {
	   operator_code = 1;
   }

   // Conver Decimal to Binary
   string x;

   x = decToBinary(input1);

   // Add the missing zeros
   x = addZeros(x ,numBits);

   for ( int i = 0; i < numBits; i++){
       pt[i] = x[i];
   }

   cout << "\nThe operator code is: " << operator_code << "\n";
    
   for (int i = 0; i < numBits; i ++) {
   	Encrypt(ct[i], pt[i], pri_key);
    }

    Synchronize();

    //-----------------------SENDING DATA OVER----------------------------

    //DUMP CTXT FILES TO SEND
    for (int i = 0; i < numBits; i ++) {
	    string filename = "cipher/ct" + std::to_string(i);
	    WriteCtxtToFile(ct[i],filename);
    }

    remove("cipher/overall");

    // Write Operator Code
    ofstream myfile;
    myfile.open ("cipher/overall");
    myfile << operator_code << "\n";
    myfile.close();

    
    for (int i = 0; i < numBits; i ++) {
	    std::ifstream if_a("cipher/ct"+std::to_string(i),std::ios_base::app);
	    std::ofstream of_c("cipher/overall",std::ios_base::app);
	    of_c << if_a.rdbuf();
    }
    
    gettimeofday(&start, NULL);
    
    // SEND INPUT AND OPERATOR CODE TO SERVER
    Client_socket C;
    C.start_everything(4386, "client");
    C.transmit_file();
    C.close_socket();

    gettimeofday(&end, NULL);
    get_time = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1.0E-6;
    printf("Time taken to send: %lf[sec]\n", get_time);

    gettimeofday(&start, NULL);

    for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

    delete [] st;

    cout<< "\n------ Cleaning Data on GPU(s) ------" <<endl;
    CleanUp(); // essential to clean and deallocate data
    delete [] ct;
    delete [] pt;
    return 0;
}

