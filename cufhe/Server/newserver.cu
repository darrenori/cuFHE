#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include<iostream>
#include<fstream>
#include<stdio.h>
#include <unistd.h>
#include <sys/socket.h>
#include <stdlib.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <strings.h>
#include <string>
#include <cmath>
#include<algorithm>
#include<iterator>


#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;


void addBits(Ctxt *r, Ctxt &a, Ctxt &b, Ctxt *carry) {
	Ctxt *t1 = new Ctxt[1];
    Ctxt *t2 = new Ctxt[1];
	Xor(t1[0], a, carry[0]);
    Xor(t2[0], b, carry[0]);
	Synchronize();
	Xor(r[0], a, t2[0]);
	And(t1[0], t1[0], t2[0]);
	Synchronize();
	Xor(r[1], carry[0], t1[0]);
	Synchronize();
	delete [] t1;
	delete [] t2;
}

void addNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
  	
	Ctxt *carry = new Ctxt[1];
        Ctxt *bitResult = new Ctxt[2];

	Xor(ctRes[nBits-1], ctA[nBits-1], ctB[nBits-1]);
	And(carry[0], ctA[nBits-1], ctB[nBits-1]);
	Synchronize();
	for(int i = nBits-2; i > 0; i--) {
		addBits(bitResult, ctA[i], ctB[i], carry);
		Copy(ctRes[i], bitResult[0]);
		Copy(carry[0], bitResult[1]);
		Synchronize();
	}
	Copy(ctRes[0], carry[0]);
	Synchronize();
	delete [] carry;
	delete [] bitResult;
}



void twoComplements(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, Ctxt *minusEnd, int nBits){
             
	    Ctxt *twoRes = new Ctxt[nBits];
	
	    // Inverse B
            for(int i = 0; i < nBits; i++){
                Not(ctB[i], ctB[i]);
            }
 
            Synchronize();
 
            // Add One to B
            addNumbers(twoRes, minusEnd, ctB, nBits);
 
            // Add result to A
            addNumbers(ctRes, ctA, twoRes, nBits);

       	    Not(ctRes[0], ctRes[0]);

	    delete [] twoRes;

};

void subNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int nBits) {
	Ctxt *minusEnd = new Ctxt[nBits];
	
	for(int i = 0; i < nBits; i ++){
	    Copy(minusEnd[i], ctA[0]);
	};

	Not(minusEnd[nBits-1], minusEnd[nBits-1]);

        twoComplements(ctRes, ctA, ctB, minusEnd, nBits);

	delete [] minusEnd;
};

void mulNumbers(Ctxt *ctRes, Ctxt *ctA, Ctxt *ctB, int iBits, int oBits){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	uint32_t kNumSMs = prop.multiProcessorCount;
	Stream* st = new Stream[kNumSMs];
	for (int i = 0; i < kNumSMs; i ++) {
		st[i].Create();
	}

	Ctxt* tempSum = new Ctxt[oBits];
	Ctxt* tempSum2 = new Ctxt[oBits];
	Ctxt* andRes = new Ctxt[iBits];
	Ctxt* empty = new Ctxt[oBits];

	//MAKE IT ZERO
	Ctxt* zero = new Ctxt[1];
	And(zero[0],ctA[0],ctB[0],st[0 % kNumSMs]);
	Xor(zero[0],zero[0],zero[0],st[0 % kNumSMs]);
	Synchronize();

	for(int i=0; i<oBits; i++){
		Copy(tempSum[i],zero[0]);
		Copy(tempSum2[i],zero[0]);
	};

	int co=0;
	int counter=0;
	Synchronize();

	for(int i = iBits-1; i > -1; i--) {
		co=0;
		co=counter;

		Ctxt* andResLeft = new Ctxt[oBits];
		//initalize nresleft to be 'nothing'
		for(int i=0; i<oBits; i++){
			Copy(andResLeft[i],zero[0]);
		}
		Synchronize();

		for(int j = 0; j < iBits; j++) {
			And(andRes[j], ctA[oBits-1-j], ctB[oBits-1-counter], st[j % kNumSMs]);
		}
		Synchronize();

		for(int j = 0; j < iBits; j++) {
			//cout << oBits-1-co;
			Copy(andResLeft[oBits-1-co], andRes[j]);
			co++;
		}
		Synchronize();


                if(counter==0) {
			addNumbers(tempSum, andResLeft, tempSum2, oBits);
		} else {
			addNumbers(tempSum, andResLeft, tempSum, oBits);
		}

		delete [] andResLeft;
		counter++;
		Synchronize();
	}

	for(int i=0; i < oBits; i ++) {
                Copy(ctRes[i], tempSum[i]);
        }
	Synchronize();
	for (int i = 0; i < kNumSMs; i ++)
		st[i].Destroy();
	delete [] st;
	delete [] tempSum;
	delete [] andRes;
}



class Server_socket{

    fstream file;
    fstream file2;

    int PORT;
    int check;

    int general_socket_descriptor;
    int new_socket_descriptor;
    struct sockaddr_in address;
    int address_length;

    public:
        Server_socket(){

	};

	void start_everything(int number, int option, string role){
            create_socket();

            PORT = number;

            cout << " The port is: " << PORT << "\n";
	    
	    address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

	    if ( role == "server" ){

            	address.sin_addr.s_addr = INADDR_ANY;
		bind_socket();
            	set_listen_set();
            	accept_connection();
	    
            } else {

                const char* array[2] ={"192.168.0.5","192.168.0.2"};

                if(inet_pton(AF_INET, array[option], &address.sin_addr)<=0) {
                      cout<<"[ERROR] : Invalid address\n";
                }

                create_connection();
	    }
        }

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) == 0) {
                perror("[ERROR] : Socket failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
	    const int enable = 1;
            if (setsockopt(general_socket_descriptor, SOL_SOCKET, SO_REUSEADDR, &enable, sizeof(int)) < 0)
                   perror("setsockopt(SO_REUSEADDR) failed");
            }  

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))!=0) {

                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) {
                sleep(10);
                create_connection();
            } else {
                cout<<"[LOG] : Connection Successfull.\n";
            }
        }

        void accept_connection(){
            if ((general_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void transmit_file(string filename){
            cout << "\nI am transmitting : " << filename << "\n";
            file.open(filename, ios::in | ios::binary);
            if(file.is_open()){
                cout<<"[LOG] : File Created.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            std::string contents((std::istreambuf_iterator<char>(file)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";
	    cout<<"===============================\n";
        }

	void close_socket(){
            close(general_socket_descriptor);
	}

	void split_file(int clientID,int numBits){
            int count = 0;
	    std::ifstream file("cipher/overall");

	    std::string filenames[numBits*2];

	    if(clientID==0) {
            	for (int i = 0; i < numBits; i ++){
                	string filename = "cipher/ct" + std::to_string(i);
			remove(filename.c_str());
			filenames[i] = filename;
	    	};
	    } else {
            	for (int i = numBits; i < numBits*2; i ++){
                	string filename = "cipher/ct" + std::to_string(i);
			remove(filename.c_str());
			filenames[i] = filename;
	    	};
	    }

	    if (file.is_open()) {
    	 	std::string line;
    		while (std::getline(file, line)) {
                      if( line.length() == 1){
		           ofstream File;
			   File.open("operator.txt", fstream::app);
			   File << line.c_str() << endl;
			   cout << "\n Reading from file is: " << line.c_str() << "\n"; 
			   File.close();
		      } else {
			   int fileChoice = floor(count/501);
			   if(clientID != 0) {
		           	fileChoice = numBits+floor(count/501);
				
		           }
	              	   ofstream Myfile;
		      	   Myfile.open(filenames[fileChoice], fstream::app);
		      	   Myfile << line.c_str() << endl;
		      	   count += 1;
		      }

	        };
	     };
        };

	void receive_file(string filename){

            file2.open(filename, ios::out | ios::trunc | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Return File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

	    char buffer[2200024] = {};
	    bzero(buffer, sizeof(buffer));
	    printf("Starting to download file contents");
	    while(1){
                    printf("Beginning file contents");
	            int valread = read(general_socket_descriptor , buffer, 2200024);
		    printf("%d",valread);
		    if(valread == 0)
			    break;
                    printf("%s", buffer);
 		    file2<<buffer;
		    bzero(buffer, sizeof(buffer));
	    };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
	    file2.close();

       };


};

int countLines(string filename) {
	ifstream aFile (filename);
        std::size_t lines_count =0;
        std::string line;
        while (std::getline(aFile, line))
                ++lines_count;
        return lines_count;

}

int main() {
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  uint32_t kNumSMs = prop.multiProcessorCount;
  uint32_t kNumLevels = 4;
  int numBits = 32;

  remove("operator.txt");
  remove("cipher/overall");
  remove("finalkeys/publickey1.txt");


  //RECEIVE OPERATOR FORM VERIF
  Server_socket O;
  O.start_everything(4380,0,"server");
  O.receive_file("operator.txt");

  //SEND OPERATOR OVER
  Server_socket T;
  T.start_everything(4381,0,"client");
  T.transmit_file("operator.txt");

  //RECEIVE KEY FROM KEYGEN
  Server_socket K;
  K.start_everything(4383,0,"server");
  K.receive_file("finalkeys/publickey1.txt");

  //RECEIVE DATA FROM CLIENT1
  Server_socket S;
  S.start_everything(4386,0,"server");
  S.receive_file("cipher/overall");


  int numLines;
  numLines=countLines("cipher/overall");
  numBits = numLines-1;
  numBits = numBits / 501;

  cout << "Nummber of bits in cipher/overall is" << numBits;


  S.split_file(0,numBits);


  //RECEIVE DATA FROM CLIENT2
  Server_socket S10;
  S10.start_everything(4387,1,"server");
  S10.receive_file("cipher/overall");
  S10.split_file(10,numBits);



  Ptxt* pt = new Ptxt[numBits];
  Ptxt* pt1 = new Ptxt[numBits];
  Ptxt* ptRes = new Ptxt[numBits*2];
  Ctxt* ct = new Ctxt[numBits];
  Ctxt* ct1 = new Ctxt[numBits];
  Ctxt* ctRes = new Ctxt[numBits*2];
  Synchronize();


  PubKey pub_key;
  ReadPubKeyFromFile(pub_key,"finalkeys/publickey1.txt");

  cout<< "------ Initilizating Data on GPU(s) ------" <<endl;
  Initialize(pub_key); // essential for GPU computing

  for (int i = 0; i < numBits; i ++) {
	  ReadCtxtFromFile(ct[i],"cipher/ct"+std::to_string(i));
  }

  int countCT=0;
  for (int i = numBits; i < numBits + numBits; i ++) {
	  ReadCtxtFromFile(ct1[countCT],"cipher/ct"+std::to_string(i));
	  countCT++;
  }


  cout<< "Number of bits:\t" << numBits <<endl;
  // Create CUDA streams for parallel gates.
  Stream* st = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i ++)
    st[i].Create();

  Synchronize();


  Ctxt* zero = new Ctxt[numBits];
  Ctxt* temp = new Ctxt[numBits];
  And(zero[numBits-1],ct[0],ct1[0],st[0 % kNumSMs]);
  Xor(zero[numBits-1],zero[0],zero[0],st[0 % kNumSMs]);
  Synchronize();

  Ctxt* one = new Ctxt[1];
  Not(one[0], zero[numBits-1]);

  for ( int i = 0; i < numBits-1; i++ ){
    Copy(zero[i], one[0]);
  };

  std::string p,q,t;
  string operators[3];
  int counter = 0;

  // Read Operator.txt
  ifstream MyReadFile("operator.txt");
  std::string myText;
  while (getline (MyReadFile, myText)) {
     operators[counter] = myText;
     counter += 1;
  }

  p = operators[1];
  q = operators[2];
  t = operators[0];
  cout << "\n The operator is " << t;
  cout << "\n Client 1 is " << p;
  cout << "\n Client 2 is " << q;
  MyReadFile.close(); 

  float et;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  //Copy(ctRes[0], one[0]);
  //for (int i=0; i < numBits; i++) 
  	//And(ctRes[i],ct[i],ct1[i],st[i % kNumSMs]);
  
  if ( (p=="1" && q=="1" && t=="1") || (p=="1" && q=="2" && t=="2") ){
      cout << "\n Adding x+y \n";
      addNumbers(ctRes, ct, ct1, numBits);
  }  else if ( (p=="1" && q=="1" && t=="2") || (p=="1" && q=="2" && t=="1") ){
      subNumbers(ctRes, ct, ct1, numBits);
      cout << "\n Subtracting x-y \n";
  }  else if ( (p=="2" && q=="1" && t=="1") || (p=="2" && q=="2" && t=="2") ){
      cout << "\n Subtracting y-x \n";
      subNumbers(ctRes, ct1, ct, numBits);
  }  else if ( (p=="2" && q=="2" && t=="1") || (p=="2" && q=="1"&& t=="2") ){
      cout << "\n Adding -x-y \n";
      addNumbers(ctRes, ct, ct1, numBits);

      for ( int i = 0; i < numBits; i++ ){
            Not(temp[i], ctRes[i]);
      };
      Synchronize();

      addNumbers(ctRes, zero, temp, numBits);
      Not(ctRes[0], ctRes[0]);
  }  else if ( (p=="1" && q=="1" && t=="3") || (p=="2" && q=="2" && t=="3") ){
      cout << "\n x*y \n";
      mulNumbers(ctRes, ct, ct1, (numBits/2), numBits);
  }  else if ( (p=="2" && q=="1" && t=="3") || (p=="1"&& q=="2" && t=="3") ){
      cout << "\n -(x*y) \n";
      mulNumbers(ctRes, ct, ct1, (numBits/2), numBits);

      for ( int i = 0; i < numBits; i++ ){
             Not(temp[i], ctRes[i]);
      };
      Synchronize();

      for ( int i = 0; i < numBits; i ++) {
              Copy(ctRes[i], temp[i]);
      }

      addNumbers(ctRes, zero, temp, numBits);
      Not(ctRes[0], ctRes[0]);
  };
  

  Synchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&et, start, stop);
  cout<< et / kNumLevels << " ms to calculate" <<endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);


  string fname;
  remove("cipherRes/overall");
  for (int i = 0; i < numBits; i ++) {
	  fname = "cipherRes/ct"+std::to_string(i);
	  remove(fname.c_str());
	  WriteCtxtToFile(ctRes[i],fname);
  }
  
  //only 32 files for cipheres
  for (int i = 0; i < numBits; i ++) {
            std::ifstream if_a("cipherRes/ct"+std::to_string(i),std::ios_base::app);
            std::ofstream of_c("cipherRes/overall",std::ios_base::app);
            of_c << if_a.rdbuf();
  }

  Server_socket S1;
  S1.start_everything(4388,1,"client");
  S1.transmit_file("cipherRes/overall");

  for (int i = 0; i < kNumSMs; i ++)
    st[i].Destroy();
  
  delete [] st;

  cout<< "------ Cleaning Data on GPU(s) ------" <<endl;
  CleanUp(); // essential to clean and deallocate data
  delete [] ct;
  delete [] pt;
  return 0;
}
