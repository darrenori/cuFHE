#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include <sys/socket.h>
#include <unistd.h>
#include <netinet/in.h>
#include <ios>
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

class Client_socket{
    fstream file;

    int PORT;
    
    int general_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:
        Client_socket(){
	
	};

	void start_everything(int number, int option, string role){
            create_socket();
            PORT = number;
	    //int new_socket_descriptor;

            address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

	    cout << "The port is " << PORT << "\n";

	    if ( role == "server" ){

                address.sin_addr.s_addr = INADDR_ANY;
                bind_socket();
                set_listen_set();
                accept_connection();

            } else {

	        const char* array[4] ={"69.69.69.2","69.69.69.1","69.69.69.3","69.69.69.4"};

            	if(inet_pton(AF_INET, array[option], &address.sin_addr)<=0) { 
                      cout<<"[ERROR] : Invalid address\n";
            	}

            	create_connection();
            
            	file.open("rec.txt", ios::out | ios::trunc | ios::binary);
            	
		if(file.is_open()){
                	cout<<"[LOG] : File Created.\n";
            	}

            	else{
                	cout<<"[ERROR] : File creation failed, Exititng.\n";
                	exit(EXIT_FAILURE);
            	}
	   }
        };

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))!=0) {

                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void accept_connection(){
            if ((general_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) < 0) {
                perror("[ERROR] : Socket failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
            const int enable = 1;
            if (setsockopt(general_socket_descriptor, SOL_SOCKET, SO_REUSEADDR, &enable, sizeof(int)) < 0)
                   perror("setsockopt(SO_REUSEADDR) failed");
        }
	
        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) { 
                //perror("[ERROR] : connection attempt failed.\n");
		sleep(10);
            	create_connection();
                //exit(EXIT_FAILURE);
            } else {
            	cout<<"[LOG] : Connection Successfull.\n";
            }
        }

        void close_socket(){
	    close(general_socket_descriptor);
	};

 
        void transmit_file(){

	    fstream file2;
            
            printf("============================\n");	    
	    file2.open("keys", ios::in | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Send File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            	    
            std::string contents((std::istreambuf_iterator<char>(file2)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";	
	}

	void receive_file(){
            remove("operator.txt");

            fstream file2;

            file2.open("operator.txt", ios::out | ios::trunc | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Return File Creted.\n";
            }
            else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            char buffer[2200024] = {};
            bzero(buffer, sizeof(buffer));
            //int count = 0;
            printf("Starting to download file contents\n");
            while(1){
                    printf("Beginning file contents\n");
                    int valread = read(general_socket_descriptor , buffer, 2200024);
                    printf("%s", buffer);
                    file2<<buffer;
		    break;
            };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
            file2.close();

          };


};




int main(int argc, char const* argv[]){
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    //uint32_t kNumSMs = prop.multiProcessorCount;

    SetSeed();
    PriKey pri_key; // private key
    PubKey pub_key;

    //Generate Temporary Keys
    KeyGen(pub_key, pri_key);

    //Write Keys to file
    WritePubKeyToFile(pub_key,"finalkeys/publickey1.txt");
    WritePriKeyToFile(pri_key,"finalkeys/privatekey1.txt");
    Initialize(pub_key);

    //Wait for operator to be sent to KeyGen
    Client_socket C0;
    C0.start_everything(4381,0,"server");
    C0.receive_file();

    //Send Private key to verif
    remove("keys");
    std::ifstream if_a("finalkeys/privatekey1.txt",std::ios_base::app);
    std::ofstream of_c("keys",std::ios_base::app);
    of_c << if_a.rdbuf();

    cout << ("\n----Sending to verif----\n");
    Client_socket C;
    C.start_everything(4382,0, "client");
    C.transmit_file();

    //Send Public key to server
    remove("keys");
    std::ifstream if_a1("finalkeys/publickey1.txt",std::ios_base::app);
    std::ofstream of_c1("keys",std::ios_base::app);
    of_c1 << if_a1.rdbuf();
    cout << ("\n----Sending to server----\n");
    Client_socket C1;
    C1.start_everything(4383,1, "client");
    C1.transmit_file();

    //Send private key to c1
    remove("keys");
    std::ifstream if_a2("finalkeys/privatekey1.txt",std::ios_base::app);
    std::ofstream of_c2("keys",std::ios_base::app);
    of_c2 << if_a2.rdbuf();
    cout << ("\n----Sending to c1----\n");
    Client_socket C2;
    C2.start_everything(4384,2, "client");
    C2.transmit_file();




    //Send private key to c2
    remove("keys");
    std::ifstream if_a3("finalkeys/privatekey1.txt",std::ios_base::app);
    std::ofstream of_c3("keys",std::ios_base::app);
    of_c3 << if_a3.rdbuf();
    cout << ("\n----Sending to c2----\n");
    Client_socket C3;
    C3.start_everything(4385,3, "client");
    C3.transmit_file();
  
}
//developer note: Did not loop for readability
