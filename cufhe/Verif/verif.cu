#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <arpa/inet.h>
#include <string.h>
#include <cmath>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
#include <netinet/in.h>

#include <ios>
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>

using namespace std;

class Client_socket{
    fstream file;

    int PORT;
    
    int general_socket_descriptor;
    int new_socket_descriptor;

    struct sockaddr_in address;
    int address_length;

    public:
        Client_socket(){
	
	};

        void start_everything(int number, string role){
            create_socket();
            PORT = number;

            cout << "The port is " << PORT << "\n";

            address.sin_family = AF_INET;
            address.sin_port = htons( PORT );
            address_length = sizeof(address);

            if ( role == "server" ){

                address.sin_addr.s_addr = INADDR_ANY;
                bind_socket();
                set_listen_set();
                accept_connection();

            } else {

                if(inet_pton(AF_INET, "192.168.0.1", &address.sin_addr)<=0) {
                        cout<<"[ERROR] : Invalid address\n";
                }

                create_connection();

                file.open("rec.txt", ios::out | ios::trunc | ios::binary);
                if(file.is_open()){
                        cout<<"[LOG] : File Creted.\n";
                }
                else{
                        cout<<"[ERROR] : File creation failed, Exititng.\n";
                        exit(EXIT_FAILURE);
                }
            }
        };

        void bind_socket(){
            if (bind(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address))!=0) {

                perror("[ERROR] : Bind failed");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Bind Successful.\n";
        }

        void set_listen_set(){
            if (listen(general_socket_descriptor, 3) < 0) {
                perror("[ERROR] : Listen");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket in Listen State (Max Connection Queue: 3)\n";
        }

        void accept_connection(){
            if ((general_socket_descriptor = accept(general_socket_descriptor, (struct sockaddr *)&address, (socklen_t*)&address_length))<0) {
                perror("[ERROR] : Accept");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Connected to Client.\n";
        }

        void create_socket(){
            if ((general_socket_descriptor = socket(AF_INET, SOCK_STREAM, 0)) < 0) { 
                perror("[ERROR] : Socket failed.\n");
                exit(EXIT_FAILURE);
            }
            cout<<"[LOG] : Socket Created Successfully.\n";
            const int enable = 1;
            if (setsockopt(general_socket_descriptor, SOL_SOCKET, SO_REUSEADDR, &enable, sizeof(int)) < 0)
                   perror("setsockopt(SO_REUSEADDR) failed");

	}

        void create_connection(){
            if (connect(general_socket_descriptor, (struct sockaddr *)&address, sizeof(address)) < 0) { 
		sleep(10);
            	create_connection();
            } else {
            	cout<<"[LOG] : Connection Successfull.\n";
            }
        }

        void close_socket(){
	    close(general_socket_descriptor);
	};

 
        void transmit_file(){

	    fstream file2;
            
            printf("============================\n");	    
	    file2.open("operator.txt", ios::in | ios::binary);
            if(file2.is_open()){
                cout<<"[LOG] : Send File is ready to Transmit.\n";
            }
            else{
                cout<<"[ERROR] : File loading failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            	    
            std::string contents((std::istreambuf_iterator<char>(file2)), std::istreambuf_iterator<char>());
            cout<<"[LOG] : Transmission Data Size "<<contents.length()<<" Bytes.\n";

            cout<<"[LOG] : Sending...\n";

            int bytes_sent = send(general_socket_descriptor , contents.c_str() , contents.length() , 0 );
            cout<<"[LOG] : Transmitted Data Size "<<bytes_sent<<" Bytes.\n";

            cout<<"[LOG] : File Transfer Complete.\n";	
	}

        void receive_file(int recvKey){
	    
            fstream file2;

	    if(recvKey==1) {
		    
		remove("finalkeys/privatekey1.txt");
                file2.open("finalkeys/privatekey1.txt", ios::out | ios::trunc | ios::binary);
		    
	    } else {
		   
		remove("cipher/overall");
            	file2.open("cipher/overall", ios::out | ios::trunc | ios::binary);
	    }
	    
            if(file2.is_open()){
               	 cout<<"[LOG] : Return File Creted.\n";
            } else{
                cout<<"[ERROR] : File creation failed, Exititng.\n";
                exit(EXIT_FAILURE);
            }

            char buffer[2200024] = {};
            bzero(buffer, sizeof(buffer));
            //int count = 0;
            printf("Starting to download file contents");
            while(1){
                   int valread = read(general_socket_descriptor , buffer, 2200024);
                   printf("%d",valread);
                   if(valread == 0)
                            break;
                   printf("%s", buffer);
                   file2<<buffer;
                   bzero(buffer, sizeof(buffer));
            };
            cout<<"[LOG] : Saving data to file.\n";
            cout<<"[LOG] : File Saved.\n";
            file2.close();
	    

        };


	void split_file(){
            int count = 0;
            std::ifstream file("cipher/overall");

            std::string filenames[32];
            for (int i = 0; i < 32; i ++){
                string filename = "cipherRes/ct" + std::to_string(i);
                remove(filename.c_str());
                filenames[i] = filename;
            };

            if (file.is_open()) {
                std::string line;
                while (std::getline(file, line)) {
                               int fileChoice = floor(count/501);
                               ofstream Myfile;
                               Myfile.open(filenames[fileChoice], fstream::app);
                               Myfile << line.c_str() << endl;
                               count += 1;
	
                };
             };
        };


};

// function to convert decimal to binary
std::string decToBinary(int n)
{
    std::string r;
    while(n!=0) {r=(n%2==0 ?"0":"1")+r; n/=2;}
    return r;
}

// function to convert binary to decimal
std::string binToDecimal(string s)
{
    unsigned long long value = std::stoull(s, 0, 2);
    std::string str = std::to_string(value);
    return str;
}

//addition
string add(string a, string b){
   string result = "";
   int temp = 0;
   int size_a = a.size() - 1;
   int size_b = b.size() - 1;
   while (size_a >= 0 || size_b >= 0 || temp == 1){
      temp += ((size_a >= 0)? a[size_a] - '0': 0);
      temp += ((size_b >= 0)? b[size_b] - '0': 0);
      result = char(temp % 2 + '0') + result;
      temp /= 2;
      size_a--; size_b--;
   }
   return result;
}

// function to convert to Two's Complement
std::string toTwoComplement(string s) {

  int size = s.length();

  char binary[size + 1], one[size + 1];
  int i;

  strcpy(binary, s.c_str());

  for (i = 0; i < size; i++) {
    if (binary[i] == '1') {
      one[i] = '0';
    } else if (binary[i] == '0') {
      one[i] = '1';
    }
  }
  one[size] = '\0';

  return add(one,"1");
}

int main(int argc, char const* argv[])
{
    int port1 = 4380;
    struct sockaddr_in;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    //uint32_t kNumLevels = 4;
    int numBits = 32;

    SetSeed();

    Ptxt* pt = new Ptxt[numBits];
    Ptxt* pt1 = new Ptxt[numBits];
    Ptxt* ptRes = new Ptxt[numBits];
    Ctxt* ct = new Ctxt[numBits];
    Ctxt* ct1 = new Ctxt[numBits];
    Ctxt* ctRes = new Ctxt[numBits];

    Stream* st = new Stream[kNumSMs];
    for (int i = 0; i < kNumSMs; i ++)
      st[i].Create();

   // Getting the User Inputs
   int operator_code;
   string sign;
   char input1;

   cout << "What is your operator: ";
   cin >> input1;


   if (input1 == '+'){
       operator_code = 1;
   } else if (input1 == '-') {
       operator_code = 2;
   } else {
       operator_code = 3;
   }

   cout << "\nThe operator code is: " << operator_code << "\n";
    
   Synchronize();

   //-----------------------SENDING DATA OVER----------------------------

   remove("operator.txt");

   // Write Operator Code
   ofstream myfile;
   myfile.open ("operator.txt");
   myfile << operator_code << "\n";
   myfile.close();
    

   //Send Operator Code to Server
   Client_socket C;
   C.start_everything(port1, "client");
   C.transmit_file();
   C.close_socket();


   //-------------------READING BACK DATA FROM SERVER----------------------//

   // Get Private Key from Key Generation
   Client_socket K;
   K.start_everything(4382, "server");
   K.receive_file(1);
   K.close_socket();
   PriKey pri_key;
   ReadPriKeyFromFile(pri_key,"finalkeys/privatekey1.txt");


   //Receive Encrypted Data from Server after is computes
   Client_socket S1;
   S1.start_everything(4388, "server");
   S1.receive_file(0);
   S1.split_file();
   S1.close_socket();

   for (int i = 0; i < numBits; i ++) {
           string filename = "cipherRes/ct" + std::to_string(i);
           ReadCtxtFromFile(ctRes[i],filename);
   }

   //Decrypt Data
   for (int i = 0; i < numBits; i ++) {
     Decrypt(ptRes[i], ctRes[i], pri_key);
   }

    
   std::string result;
   //Print out result
   cout << "\nRESULT:\n";
   for (int i=0; i < numBits; i++) {
	   result = result + std::to_string(ptRes[i].message_);
   }

   cout << "\n The result is : " << result;
   if(result[0] == '1'){
	result = toTwoComplement(result);
	result = "-" + binToDecimal(result);
   } else {
       result = binToDecimal(result);
   };

   cout << "\n The result is: " << result;
    
   for (int i = 0; i < kNumSMs; i ++)
      st[i].Destroy();

   delete [] st;

   cout<< "\n------ Cleaning Data on GPU ------\n";
   CleanUp(); // essential to clean and deallocate data
   delete [] ct;
   delete [] pt;
   return 0;
}
