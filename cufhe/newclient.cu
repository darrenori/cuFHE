// Client side C/C++ program to demonstrate Socket
// programming
#include <arpa/inet.h>
#include <stdio.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>

#define PORT 4380
  
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;


int main(int argc, char const* argv[])
{
    int sock = 0, valread, client_fd;
    struct sockaddr_in serv_addr;

    // Generating CT

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    uint32_t kNumTests = kNumSMs * 32;// * 8;
    uint32_t kNumLevels = 4;

    SetSeed(); // set random seed

    PriKey pri_key; // private key
    PubKey pub_key; // public key
    Ptxt* pt = new Ptxt[2 * kNumTests];
    Ctxt* ct = new Ctxt[2 * kNumTests];
    Synchronize();

    // End of Generation of CT

    char* hello = "Hello from client";
    char buffer[1024] = { 0 };
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0) {
        printf("\n Socket creation error \n");
        return -1;
    }
  
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_port = htons(PORT);
  
    // Convert IPv4 and IPv6 addresses from text to binary
    // form
    if (inet_pton(AF_INET, "127.0.0.1", &serv_addr.sin_addr)
        <= 0) {
        printf(
            "\nInvalid address/ Address not supported \n");
        return -1;
    }
  
    if ((client_fd
         = connect(sock, (struct sockaddr*)&serv_addr,
                   sizeof(serv_addr)))
        < 0) {
        printf("\nConnection Failed \n");
        return -1;
    }
    send(sock, ct, strlen(ct), 0);
    printf("Hello message sent\n");
    valread = read(sock, buffer, 1024);
    printf("%s\n", buffer);
  
    // closing the connected socket
    close(client_fd);
    return 0;
}
